#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <math.h>

#define N 1000
#define tau 0.01
#define max_pos 5.0
#define min_pos -5.0

char fname[] = "results[n = 1000].txt";

const double tmax = 1.0;
double t = 0.0;

using namespace std;

__global__ void Kernel(double* U, double* U_new) {

	int A1 = 1, A2 = 2;
	int p1 = 3, p2 = 2;

	double sum_Vx = 0.0, sum_Vy = 0.0;

	__shared__ double temp[4 * N];

	int i = blockIdx.x * blockDim.x * 4 + threadIdx.x * 4;

	if (i < N * 4)
	{
		temp[i + 2] = U[i + 2];
		temp[i + 3] = U[i + 3];
	}

	__syncthreads();

	if (i < N * 4)
	{
		for (int j = 0; j < N * 4; j += 4)
		{
			if (i != j)
			{

				temp[i] = (A1 * (temp[j + 2] - temp[i + 2])) / pow(sqrt(pow(temp[j + 2] - temp[i + 2], 2) + pow(temp[j + 3] - temp[i + 3], 2)), p1) -
					(A2 * (temp[j + 2] - temp[i + 2])) / pow(sqrt(pow(temp[j + 2] - temp[i + 2], 2) + pow(temp[j + 3] - temp[i + 3], 2)), p2);

				temp[i + 1] = (A1 * (temp[j + 3] - temp[i + 3])) / pow(sqrt(pow(temp[j + 2] - temp[i + 2], 2) + pow(temp[j + 3] - temp[i + 3], 2)), p1) -
					(A2 * (temp[j + 3] - temp[i + 3])) / pow(sqrt(pow(temp[j + 2] - temp[i + 2], 2) + pow(temp[j + 3] - temp[i + 3], 2)), p2);

				sum_Vx += temp[i];
				sum_Vy += temp[i + 1];

				U_new[i] = temp[i] + tau * sum_Vx;
				U_new[i + 1] = temp[i + 1] + tau * sum_Vy;

				double tmp_x = temp[i + 2] + tau * U_new[i];
				double tmp_y = temp[i + 3] + tau * U_new[i + 1];

				if ((tmp_x > max_pos) || (tmp_x < min_pos))
				{
					U_new[i] = -U_new[i];
					U_new[i + 2] = temp[i + 2] + tau * U_new[i];
				}
				else
				{
					U_new[i + 2] = tmp_x;
				}

				if ((tmp_y > max_pos) || (tmp_y < min_pos))
				{
					U_new[i + 1] = -U_new[i + 1];
					U_new[i + 3] = temp[i + 3] + tau * U_new[i + 1];
				}
				else
				{
					U_new[i + 3] = tmp_y;
				}
			}
		}

		temp[i] = U_new[i];
		temp[i + 1] = U_new[i + 1];
		temp[i + 2] = U_new[i + 2];
		temp[i + 3] = U_new[i + 3];
	}
}

int main() {

	ofstream file(fname);

	if (file.is_open()) {

		hipEvent_t tn, tk;
		float runtime = 0.0;

		int size = sizeof(double) * 4 * N;

		double* U = new double[4 * N];

		for (int i = 0; i < (4 * N); i += 4)
		{
			U[i + 2] = (double)(rand()) / RAND_MAX - 0.5;
			U[i + 3] = (double)(rand()) / RAND_MAX - 0.5;
		}

		double* Unew_Dev = NULL;
		double* U_Dev = NULL;

		hipMalloc((void**)&Unew_Dev, size);
		hipMalloc((void**)&U_Dev, size);

		hipEventCreate(&tn);
		hipEventCreate(&tk);

		hipEventRecord(tn, 0);

		hipMemcpy(U_Dev, U, size, hipMemcpyHostToDevice);

		do {

			Kernel << < 1, N >> > (U_Dev, Unew_Dev);

			hipDeviceSynchronize();

			hipMemcpy(U, Unew_Dev, size, hipMemcpyDeviceToHost);

			file << "t = " << t << endl;
			for (int i = 0; i < (4 * N); i += 4)
			{
				file << "\tU[" << U[i + 2] << ", " << U[i + 3] << "]" << endl;
			}

			t += tau;

		} while (t < tmax);

		hipEventRecord(tk, 0);
		hipEventSynchronize(tk);
		hipEventElapsedTime(&runtime, tn, tk);

		file << endl << "runtime = " << runtime / 1000.0 << " sec" << endl;

		delete[] U;
		hipFree(Unew_Dev);
		hipFree(U_Dev);
	}
	else {
		cout << "can't open file " << fname << endl;
	}

	return 0;
}